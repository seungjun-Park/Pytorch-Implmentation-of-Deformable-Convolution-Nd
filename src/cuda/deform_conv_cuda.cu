#include <torch/extension.h>

#include <cuda/im2col_cuda.h>
#include <cuda/col2im_cuda.h>
#include <hip/hip_runtime.h>
#include <c10/cuda/CUDAStream.h>

#include <GPUInfo.h>
#include <deform_conv_utils.h>
#include <array_utils.h>
#include <type_utils.h>

template<int8_t dim>
at::Tensor deform_conv_nd_forward_cuda(
	const at::Tensor& input,
	const at::Tensor& weight,
	const at::Tensor& offset_field,
	const at::Tensor& attn_mask,
	at::IntArrayRef kernel_size,
	at::IntArrayRef stride,
	at::IntArrayRef padding,
	at::IntArrayRef dilation,
	const int64_t groups,
	const int64_t deformable_groups,
	const at::Tensor& bias) {

	auto k = weight.dim();
	int64_t tensor_dim = k - 2;

	TORCH_CHECK(dim == tensor_dim);

	at::Tensor undefined;

	check_deform_conv_backend(
		input,
		weight,
		offset_field,
		attn_mask,
		bias,
		undefined,
		at::Backend::CUDA
	);

	at::Tensor output = at::zeros(get_output_size<dim>(input, weight, kernel_size, stride, padding, dilation), input.options().memory_format(at::MemoryFormat::Contiguous));

	// slice tensor sizes (b, c, *) to (*) 
	auto input_size = input.sizes();
	auto output_size = output.sizes();

	int32_t batch_size = input.size(0);
	int32_t in_channels = input.size(1);
	int32_t out_channels = weight.size(0);
	int32_t grouped_in_channels = in_channels / groups;
	int32_t grouped_out_channels = out_channels / groups;

	int32_t kernel_sizes = c10::multiply_integers(kernel_size);
	int32_t output_sizes = c10::multiply_integers(output_size.slice(2));

	torch::Device device = input.device();

	TORCH_CHECK(device.index() < torch::cuda::device_count());

	// custom class to check current gpu status.

	GPUInfo gpu_info;
	auto device_properties = gpu_info.GetDeviceProps()[device.index()];

	int64_t columns_numel = groups * kernel_sizes * grouped_in_channels * batch_size * output_sizes;
	int64_t per_elements_in_batch = groups * kernel_sizes * grouped_in_channels * output_sizes;

	int32_t min_grid_size, block_size;
	AT_DISPATCH_FLOATING_TYPES_AND2(at::kHalf, at::kBFloat16, input.scalar_type(), "get_blocks", [&]() {
		using scalar_t = scalar_t;
		hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size, im2col_nd_cuda<scalar_t, dim>, 0, device_properties.maxThreadsPerBlock);
	});

	int32_t num_blocks = (columns_numel + block_size - 1) / block_size;

	int32_t sub_batch_size = (num_blocks * block_size) / per_elements_in_batch;
	int32_t total_iteration = batch_size / sub_batch_size;

	at::Tensor columns = at::zeros({ groups, kernel_sizes * grouped_in_channels, sub_batch_size * output_sizes }, input.options().memory_format(at::MemoryFormat::Contiguous));

	std::vector<int64_t> output_n_size(2 + dim);
	output_n_size[0] = out_channels;
	output_n_size[1] = sub_batch_size;
	for (int8_t i = 0; i < dim; i++)
	{
		output_n_size[2 + i] = output_size.slice(2)[i];
	}

	auto cudaStream = c10::cuda::getCurrentCUDAStream(device.index());

	AT_DISPATCH_FLOATING_TYPES_AND2(at::kHalf, at::kBFloat16, input.scalar_type(), "deform_conv_nd_forward<>", [&]() {
		using scalar_t = scalar_t;

		for (const auto n : c10::irange(total_iteration))
		{
			int64_t batch_start = sub_batch_size * n;
			at::Tensor input_n = input.slice(0, batch_start, batch_start + sub_batch_size);
			at::Tensor offset_field_n = offset_field.slice(0, batch_start, batch_start + sub_batch_size);
			at::Tensor attn_mask_n = attn_mask.slice(0, batch_start, batch_start + sub_batch_size);

			im2col_nd_cuda<scalar_t, dim><<<num_blocks, block_size, 0, cudaStream>>>(
				input_n.const_data_ptr<scalar_t>(),
				offset_field_n.const_data_ptr<scalar_t>(),
				attn_mask_n.const_data_ptr<scalar_t>(),
				sub_batch_size,
				grouped_in_channels,
				IntArrayRef2IntArray<dim>(input_size.slice(2)),
				IntArrayRef2IntArray<dim>(output_size.slice(2)),
				IntArrayRef2IntArray<dim>(kernel_size),
				IntArrayRef2IntArray<dim>(stride),
				IntArrayRef2IntArray<dim>(padding),
				IntArrayRef2IntArray<dim>(dilation),
				groups,
				deformable_groups,
				columns.mutable_data_ptr<scalar_t>()
			);

			output.slice(0, batch_start, batch_start + sub_batch_size) = torch::bmm(
				weight.reshape({ groups, grouped_out_channels, -1 }),
				columns
			).reshape(output_n_size).transpose(0, 1);
		}
		
		if (bias.defined())
		{
			output = (output.reshape({ batch_size, out_channels, -1 }).transpose(1, 2) + bias).transpose(1, 2).reshape(output_size);
		}
		});

	return output;
}

template<int8_t dim>
torch::autograd::tensor_list deform_conv_nd_backward_cuda(
	const at::Tensor& input,
	const at::Tensor& weight,
	const at::Tensor& offset_field,
	const at::Tensor& attn_mask,
	const at::Tensor& grad_output,
	at::IntArrayRef kernel_size,
	at::IntArrayRef stride,
	at::IntArrayRef padding,
	at::IntArrayRef dilation,
	const int64_t groups,
	const int64_t deformable_groups,
	const at::Tensor& bias) {

	auto k = weight.dim();
	int64_t tensor_dim = k - 2;

	TORCH_CHECK(dim == tensor_dim);

	check_deform_conv_backend(
		input,
		weight,
		offset_field,
		attn_mask,
		bias,
		grad_output,
		at::Backend::CUDA
	);

	at::Tensor output = at::zeros(get_output_size<dim>(input, weight, kernel_size, stride, padding, dilation), input.options().memory_format(at::MemoryFormat::Contiguous));

	at::Tensor grad_input = at::zeros_like(input);
	at::Tensor grad_weight = at::zeros_like(weight);
	at::Tensor grad_offset_field = at::zeros_like(offset_field);
	at::Tensor grad_attn_mask = at::zeros_like(attn_mask);
	at::Tensor grad_bias = bias.defined() ? at::zeros_like(bias) : at::Tensor();

	// slice tensor sizes (b, c, *) to (*) 
	auto input_size = input.sizes();
	auto output_size = output.sizes();

	int64_t batch_size = input.size(0);
	int64_t in_channels = input.size(1);
	int64_t out_channels = weight.size(0);
	int64_t grouped_in_channels = in_channels / groups;
	int64_t grouped_out_channels = out_channels / groups;

	int64_t kernel_sizes = c10::multiply_integers(kernel_size);
	int64_t output_sizes = c10::multiply_integers(output_size.slice(2));

	torch::Device device = input.device();

	TORCH_CHECK(device.index() < torch::cuda::device_count());

	GPUInfo gpu_info;
	auto device_properties = gpu_info.GetDeviceProps()[device.index()];

	int64_t columns_numel = groups * kernel_sizes * grouped_in_channels * batch_size * output_sizes;
	int64_t per_elements_in_batch = groups * kernel_sizes * grouped_in_channels * output_sizes;

	int32_t min_grid_size, block_size;
	AT_DISPATCH_FLOATING_TYPES_AND2(at::kHalf, at::kBFloat16, input.scalar_type(), "get_blocks", [&]() {
		using scalar_t = scalar_t;
		int32_t min_grid_size_im2col, block_size_im2col;
		int32_t min_grid_size_col2im, block_size_col2im;
		hipOccupancyMaxPotentialBlockSize(&min_grid_size_im2col, &block_size_im2col, im2col_nd_cuda<scalar_t, dim>, 0, device_properties.maxThreadsPerBlock);
		hipOccupancyMaxPotentialBlockSize(&min_grid_size_col2im, &block_size_col2im, col2im_nd_cuda<scalar_t, dim>, 0, device_properties.maxThreadsPerBlock);
		if (block_size_col2im < block_size_im2col)
		{
			min_grid_size = min_grid_size_col2im;
			block_size = block_size_col2im;
		}
		else
		{
			min_grid_size = min_grid_size_im2col;
			block_size = block_size_im2col;
		}
		});
	int32_t num_blocks = (columns_numel + block_size - 1) / block_size;

	int64_t sub_batch_size = (num_blocks * block_size) / per_elements_in_batch;
	int64_t total_iteration = batch_size / sub_batch_size;

	auto cudaStream = c10::cuda::getCurrentCUDAStream(device.index());

	AT_DISPATCH_FLOATING_TYPES_AND2(at::kHalf, at::kBFloat16, input.scalar_type(), "deform_conv_nd_backward<>", [&]() {
		using scalar_t = scalar_t;

		for (const auto n : c10::irange(total_iteration))
		{
			int64_t batch_start = sub_batch_size * n;

			at::Tensor input_n = input.slice(0, batch_start, batch_start + sub_batch_size);
			at::Tensor offset_field_n = offset_field.slice(0, batch_start, batch_start + sub_batch_size);
			at::Tensor attn_mask_n = attn_mask.slice(0, batch_start, batch_start + sub_batch_size);

			at::Tensor grad_input_n = grad_input.slice(0, batch_start, batch_start + sub_batch_size);
			at::Tensor grad_offset_field_n = grad_offset_field.slice(0, batch_start, batch_start + sub_batch_size);
			at::Tensor grad_attn_mask_n = grad_attn_mask.slice(0, batch_start, batch_start + sub_batch_size);
			at::Tensor grad_output_n = grad_output.slice(0, batch_start, batch_start + sub_batch_size);

			at::Tensor columns = torch::bmm(
				weight.reshape({ groups, grouped_out_channels, -1 }).transpose(1, 2), 
				grad_output_n.transpose(0, 1).reshape({ groups, grouped_out_channels, -1 })
			);

			// compute gradient of inputs, offset_field, attn_mask
			col2im_nd_cuda<scalar_t, dim> << <num_blocks, block_size, 0, cudaStream>>> (
				input_n.const_data_ptr<scalar_t>(),
				columns.const_data_ptr<scalar_t>(),
				offset_field_n.const_data_ptr<scalar_t>(),
				attn_mask_n.const_data_ptr<scalar_t>(),
				sub_batch_size,
				grouped_in_channels,
				IntArrayRef2IntArray<dim>(input_size.slice(2)),
				IntArrayRef2IntArray<dim>(output_size.slice(2)),
				IntArrayRef2IntArray<dim>(kernel_size),
				IntArrayRef2IntArray<dim>(stride),
				IntArrayRef2IntArray<dim>(padding),
				IntArrayRef2IntArray<dim>(dilation),
				groups,
				deformable_groups,
				(mapped_type<scalar_t>*)grad_input_n.mutable_data_ptr<scalar_t>(),
				(mapped_type<scalar_t>*)grad_offset_field_n.mutable_data_ptr<scalar_t>(),
				(mapped_type<scalar_t>*)grad_attn_mask_n.mutable_data_ptr<scalar_t>()
				);

			// compute grad_weight = grad_output * col^T
			im2col_nd_cuda<scalar_t, dim><<<num_blocks, block_size, 0, cudaStream>>>(
				input_n.const_data_ptr<scalar_t>(),
				offset_field_n.const_data_ptr<scalar_t>(),
				attn_mask_n.const_data_ptr<scalar_t>(),
				sub_batch_size,
				grouped_in_channels,
				IntArrayRef2IntArray<dim>(input_size.slice(2)),
				IntArrayRef2IntArray<dim>(output_size.slice(2)),
				IntArrayRef2IntArray<dim>(kernel_size),
				IntArrayRef2IntArray<dim>(stride),
				IntArrayRef2IntArray<dim>(padding),
				IntArrayRef2IntArray<dim>(dilation),
				groups,
				deformable_groups,
				columns.mutable_data_ptr<scalar_t>()
			);

			// compute grad_weight = grad_output * col^T
			grad_weight += torch::bmm(
				grad_output_n.transpose(0, 1).reshape({groups, grouped_out_channels, -1}),
				columns.transpose(1, 2)
			).reshape(grad_weight.sizes());
		}

		if (grad_bias.defined())
		{
			std::vector<int64_t> dims(dim + 1, 0);
			std::iota(dims.begin() + 1, dims.end(), 2);
			grad_bias = grad_output.sum(dims);
		}
	});

	at::Tensor undefined;

	return {
		grad_input, grad_weight, grad_offset_field, grad_attn_mask,
		undefined, undefined, undefined, undefined, undefined, undefined,
		grad_bias
	};
}

TORCH_LIBRARY_IMPL(custom_op, CUDA, m)
{
	m.impl("deform_conv1d_forward", &deform_conv_nd_forward_cuda<1>);
	m.impl("deform_conv2d_forward", &deform_conv_nd_forward_cuda<2>);
	m.impl("deform_conv3d_forward", &deform_conv_nd_forward_cuda<3>);

	m.impl("deform_conv1d_backward", &deform_conv_nd_backward_cuda<1>);
	m.impl("deform_conv2d_backward", &deform_conv_nd_backward_cuda<2>);
	m.impl("deform_conv3d_backward", &deform_conv_nd_backward_cuda<3>);
}